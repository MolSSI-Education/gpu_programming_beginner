/*================================================*/
/*================ gpuVectorSum.cu ===============*/
/*================================================*/
// #include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

/*************************************************/
inline double chronometer() {
    struct timezone tzp;
    struct timeval tp;
    int tmp = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}
/*-----------------------------------------------*/
void dataInitializer(float *inputArray, int size) {
    /* Generating float-type random numbers 
     * between 0.0 and 1.0
     */
    time_t t;
    srand( (unsigned int) time(&t) );

    for (int i = 0; i < size; i++) {
        inputArray[i] = ( (float)rand() / (float)(RAND_MAX) ) * 1.0;
    }

    return;
}
/*-----------------------------------------------*/
void arraySumOnHost(float *A, float *B, float *C, const int size) {
    for (int i = 0; i < size; i++) {
        C[i] = A[i] + B[i];
    }
}
/*-----------------------------------------------*/
__global__ void arraySumOnDevice(float *A, float *B, float *C, const int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) { 
        C[idx] = A[idx] + B[idx];
    }
}
/*-----------------------------------------------*/
void arrayEqualityCheck(float *hostPtr, float *devicePtr, const int size) {
    double tolerance = 1.0E-8;
    bool isEqual = true;

    for (int i = 0; i < size; i++) {
        if (abs(hostPtr[i] - devicePtr[i]) > tolerance) {
            isEqual = false;
            std::cout << "Arrays are NOT equal because:" << std::endl;
            std::cout << "at " << i << "th index: hostPtr[" << i << "] = " <<
            std::setprecision(16) << hostPtr[i] << " and devicePtr[" << i << "] = "
            << devicePtr[i] << std::endl;
            break;
        }
    }

    if (isEqual) {
        std::cout << "Arrays are equal.\n" << std::endl;
    }
}
/*************************************************/
int main(int argc, char **argv) {
    std::cout << "Kicking off " 
    << argv[0] << "\n" << std::endl;

    /* Device setup */
    int deviceIdx = 0;
    hipSetDevice(deviceIdx);

    /* Device properties */
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    std::cout << "GPU device " << deviceProp.name <<
    " with index (" << deviceIdx << ") is set!\n" << 
    std::endl;
/*-----------------------------------------------*/
    /* Fixing the vector size to 1 * 2^24 = 16777216 (64 MB) */
    int vecSize = 1 << 24;
    size_t vecSizeInBytes = vecSize * sizeof(float);
    std::cout << "Vector size: " << vecSize << 
    " floats (" << vecSizeInBytes/1024/1024 << " MB)\n" << 
    std::endl;

    /* Memory allocation on the host */
    float *h_A  = new float[vecSizeInBytes];
    float *h_B  = new float[vecSizeInBytes];
    float *hostPtr   = new float[vecSizeInBytes]();
    float *devicePtr = new float[vecSizeInBytes]();

    double tStart, tElapsed;

    /* Vector initialization on the host */
    tStart = chronometer();
    dataInitializer(h_A, vecSize);
    dataInitializer(h_B, vecSize);
    tElapsed = chronometer() - tStart;
    std::cout << "Elapsed time for dataInitializer: "
    << tElapsed <<  " second(s)" << std::endl;

    /* Vector summation on the host */
    tStart = chronometer();
    arraySumOnHost(h_A, h_B, hostPtr, vecSize);
    tElapsed = chronometer() - tStart;
    std::cout << "Elapsed time for arraySumOnHost: "
    << tElapsed <<" second(s)" << std::endl;
/*-----------------------------------------------*/
    /* (Global) memory allocation on the device */
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, vecSizeInBytes);
    hipMalloc((float**)&d_B, vecSizeInBytes);
    hipMalloc((float**)&d_C, vecSizeInBytes);

    /* Data transfer from host to device */
    hipMemcpy(d_A, h_A, vecSizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, vecSizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, devicePtr, vecSizeInBytes, hipMemcpyHostToDevice);

    /* Organizing grids and blocks */
    int numThreadsInBlocks = 1024;
    dim3 block (numThreadsInBlocks);
    dim3 grid  ((vecSize + block.x - 1) / block.x);

    /* Execute the kernel from the host*/
    tStart = chronometer();
    arraySumOnDevice<<<grid, block>>>(d_A, d_B, d_C, vecSize);
    hipDeviceSynchronize();
    tElapsed = chronometer() - tStart;
    std::cout << "Elapsed time for arraySumOnDevice <<< "
    << grid.x << "," << block.x << " >>>: " << tElapsed 
    << " second(s)\n" << std::endl;
/*-----------------------------------------------*/
    /* Returning the last error from a runtime call */
    hipGetLastError();

    /* Data transfer back from device to host */
    hipMemcpy(devicePtr, d_C, vecSizeInBytes, hipMemcpyDeviceToHost);

    /* Check to see if the array summations on 
     * CPU and GPU yield the same results 
     */
    arrayEqualityCheck(hostPtr, devicePtr, vecSize);
/*-----------------------------------------------*/
    /* Free the allocated memory on the device */
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    /* Free the allocated memory on the host */
    delete [] h_A;
    delete [] h_B;
    delete [] hostPtr;
    delete [] devicePtr;

    return(0);
}