/*================================================*/
/*================ gpuVectorSum.cu ===============*/
/*================================================*/
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cudaCode.h"
extern "C" {
    #include "cCode.h"
}

/*************************************************/
int main(int argc, char **argv) {
    printf("Kicking off %s\n\n", argv[0]);

    /* Device setup */
    int deviceIdx = 0;
    ERRORHANDLER(hipSetDevice(deviceIdx));

    /* Device properties */
    deviceProperties(deviceIdx);
/*-----------------------------------------------*/
    /* Fixing the vector size to 1 * 2^24 = 16777216 (64 MB) */
    int vecSize = 1 << 24;
    size_t vecSizeInBytes = vecSize * sizeof(float);
    printf("Vector size: %d floats (%lu MB)\n\n", vecSize, vecSizeInBytes/1024/1024);

    /* Memory allocation on the host */
    float *h_A, *h_B, *hostPtr, *devicePtr;
    h_A     = (float *)malloc(vecSizeInBytes);
    h_B     = (float *)malloc(vecSizeInBytes);
    hostPtr = (float *)malloc(vecSizeInBytes);
    devicePtr  = (float *)malloc(vecSizeInBytes);

    double tStart, tElapsed;

    /* Vector initialization on the host */
    tStart = chronometer();
    dataInitializer(h_A, vecSize);
    dataInitializer(h_B, vecSize);
    tElapsed = chronometer() - tStart;
    printf("Elapsed time for dataInitializer: %f second(s)\n", tElapsed);
    memset(hostPtr, 0, vecSizeInBytes);
    memset(devicePtr,  0, vecSizeInBytes);

    /* Vector summation on the host */
    tStart = chronometer();
    arraySumOnHost(h_A, h_B, hostPtr, vecSize);
    tElapsed = chronometer() - tStart;
    printf("Elapsed time for arraySumOnHost: %f second(s)\n", tElapsed);
/*-----------------------------------------------*/
    /* (Global) memory allocation on the device */
    float *d_A, *d_B, *d_C;
    ERRORHANDLER(hipMalloc((float**)&d_A, vecSizeInBytes));
    ERRORHANDLER(hipMalloc((float**)&d_B, vecSizeInBytes));
    ERRORHANDLER(hipMalloc((float**)&d_C, vecSizeInBytes));

    /* Data transfer from host to device */
    ERRORHANDLER(hipMemcpy(d_A, h_A, vecSizeInBytes, hipMemcpyHostToDevice));
    ERRORHANDLER(hipMemcpy(d_B, h_B, vecSizeInBytes, hipMemcpyHostToDevice));
    ERRORHANDLER(hipMemcpy(d_C, devicePtr, vecSizeInBytes, hipMemcpyHostToDevice));

    /* Organizing grids and blocks */
    int numThreadsInBlocks = 1024;
    dim3 block (numThreadsInBlocks);
    dim3 grid  ((vecSize + block.x - 1) / block.x);

    /* Execute the kernel from the host*/
    tStart = chronometer();
    arraySumOnDevice<<<grid, block>>>(d_A, d_B, d_C, vecSize);
    ERRORHANDLER(hipDeviceSynchronize());
    tElapsed = chronometer() - tStart;
    printf("Elapsed time for arraySumOnDevice <<< %d, %d >>>: %f second(s) \n\n", \
    grid.x, block.x, tElapsed);
/*-----------------------------------------------*/
    /* Returning the last error from a runtime call */
    ERRORHANDLER(hipGetLastError());

    /* Data transfer back from device to host */
    ERRORHANDLER(hipMemcpy(devicePtr, d_C, vecSizeInBytes, hipMemcpyDeviceToHost));

    /* Check to see if the array summations on 
     * CPU and GPU yield the same results 
     */
    arrayEqualityCheck(hostPtr, devicePtr, vecSize);
/*-----------------------------------------------*/
    /* Free the allocated memory on the device */
    ERRORHANDLER(hipFree(d_A));
    ERRORHANDLER(hipFree(d_B));
    ERRORHANDLER(hipFree(d_C));

    /* Free the allocated memory on the host */
    free(h_A);
    free(h_B);
    free(hostPtr);
    free(devicePtr);

    return(EXIT_SUCCESS);
}
